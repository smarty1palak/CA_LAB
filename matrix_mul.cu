#include<stdio.h>
#include<hip/hip_runtime.h>

# define M 1000
# define N 1000

__global__ void mult( int * a, int * b, int * c)
{
	unsigned int i= blockDim.x *blockIdx.x + threadIdx.x;
	unsigned int j= blockDim.y *blockIdx.y + threadIdx.y;
	int sum=0;
	if(i<M && j<N)
	{
		for(int k=0;k<N;k++)
		{
			sum+=(a[i*N+k]* b[k*N+j]);			
		}
		c[i*N+j]=sum;
	}
	else
           return ;
}

int check(int *a, int *b, int *c)
{
	for(int i=0;i<M;i++)
	{
		for(int j=0;j<N;j++)
		{
			int sum=0;
			for(int k=0;k<N;k++)
			{
				sum+=a[i*N+k]*b[k*N+j];
			}
			if(c[i*N+j]!=sum)
				return 0;
		}
	}
	return 1;
}

int main()
{
	int *h_a, *h_b, *h_c;
	int *d_a, *d_b, *d_c;

	// allocating memory on host	
	h_a = (int *)malloc(M * N * sizeof(int));
	h_b = (int *)malloc(M * N * sizeof(int));
	h_c = (int *)malloc(M * N * sizeof(int));
	
	//assigning random values to the array elements
	for(int i=0;i<M;i++)
	{
		for(int j=0;j<N;j++)
		{
			h_a[i*M+j]=1;
			h_b[i*M+j]=2;
		}
		
	}

	
	//assigning memory on the device	
	hipMalloc((void **)&d_a, M*N*sizeof(int));
	hipMalloc((void **)&d_b, M*N*sizeof(int));
	hipMalloc((void **)&d_c, M*N*sizeof(int));

	//copying elements from host to device
	hipMemcpy(d_a, h_a, M*N*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, M*N*sizeof(int), hipMemcpyHostToDevice);


	//declaring the number of blocks and number of threads per block
	dim3 threads(32,32);
	dim3 blocks(M/32+1, N/32+1);

	//calling the function and calculating the sum on device
	mult<<< blocks, threads >>>(d_a, d_b, d_c);

	//copying the result to host memory
	hipMemcpy(h_c, d_c, M*N*sizeof(int), hipMemcpyDeviceToHost);

	if(check(h_a, h_b, h_c))
		printf("Matrix multiplication is correct\n");
	else
		printf("Matrix multiplication is incorrect\n");

	hipFree(d_a);
  	hipFree(d_b);
  	hipFree(d_c);

  	free(h_a);
  	free(h_b);
  	free(h_c);
	
}
