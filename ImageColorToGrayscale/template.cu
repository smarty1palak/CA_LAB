#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#include "wb.h"

//@@ INSERT CODE HERE

__global__ void rgb2gray (float * input, float *output, int height, int width)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x<height && y<width)
    {
      unsigned int idx = x* width + y;
      float r          = input[3 * idx];     
      float g          = input[3 * idx + 1]; // green value for pixel
      float b          = input[3 * idx + 2];
      output[idx] = (0.21f * r + 0.71f * g + 0.07f * b);
    }
}
int main(int argc, char *argv[]) {

  int imageChannels;
  int imageWidth;
  int imageHeight;
  char *inputImageFile;
  wbArg_t args;
  wbImage_t inputImage;
  wbImage_t outputImage;
  float *hostInputImageData;
  float *hostOutputImageData;
  float *deviceInputImageData;
  float *deviceOutputImageData;

  /* parse the input arguments */
  //@@ Insert code here

  args = wbArg_read(argc, argv);

  inputImageFile = wbArg_getInputFile(args, 0);

  inputImage = wbImport(inputImageFile);

  imageWidth  = wbImage_getWidth(inputImage);
  imageHeight = wbImage_getHeight(inputImage);
  // For this lab the value is always 3
  imageChannels = wbImage_getChannels(inputImage);

  // Since the image is monochromatic, it only contains one channel
  outputImage = wbImage_new(imageWidth, imageHeight, 1);

  hostInputImageData  = wbImage_getData(inputImage);
  hostOutputImageData = wbImage_getData(outputImage);

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");

  wbTime_start(GPU, "Doing GPU memory allocation");
  hipMalloc((void **)&deviceInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float));
  hipMalloc((void **)&deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float));
  wbTime_stop(GPU, "Doing GPU memory allocation");

  wbTime_start(Copy, "Copying data to the GPU");
  hipMemcpy(deviceInputImageData, hostInputImageData,
             imageWidth * imageHeight * imageChannels * sizeof(float),
             hipMemcpyHostToDevice);
  wbTime_stop(Copy, "Copying data to the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ INSERT CODE HERE
  dim3 block(32,32,1);
  dim3 grid(imageHeight/32+1, imageWidth/32+1,1);
  rgb2gray<<< grid, block >>> (deviceInputImageData, deviceOutputImageData, imageHeight, imageWidth);

  wbTime_stop(Compute, "Doing the computation on the GPU");

  ///////////////////////////////////////////////////////
  wbTime_start(Copy, "Copying data from the GPU");
  hipMemcpy(hostOutputImageData, deviceOutputImageData,
             imageWidth * imageHeight * sizeof(float),
             hipMemcpyDeviceToHost);
  wbTime_stop(Copy, "Copying data from the GPU");

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, outputImage);

  hipFree(deviceInputImageData);
  hipFree(deviceOutputImageData);

  wbImage_delete(outputImage);
  wbImage_delete(inputImage);

  return 0;
}
